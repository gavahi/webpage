#include "hip/hip_runtime.h"
#include <stdio.h>
#define N 1024 

__global__ void add( /* arguments */ ) {
	// addressing data by thread index:
	// use blockIdx.x , blockDim.x ,threadIdx.x variables

	
	// add operation
	
}

int main( void ) {

	// initiate variables for CPU: use pointer type


	
	// initiate variables for GPU: use pointer type



	// allocate the memory on the CPU:
	// <var> = (<type> *) malloc (<size> * sizeof(<type>));


	
	// allocate the memory on the GPU
	// hipMalloc( &<var>  ,  <size> * sizeof( <type> ) ) ;


	
	// fill the input arrays on the CPU
	// use "for_loop" command	
	for (int i = 0 ;  i < N ;  i++ ) {
		a[i] = i ;
		b[i] = i * i ;	}
		
	// copy the input arrays to the GPU
	// hipMemcpy( <dest_var> , <source_var> , <size> * sizeof( <type> ) , <direction> );
	
	

	// call kernel: 128 thread in each block
	// <kernel_name> <<< <# block> , <# thread> >>> ( <Argoments> );

	
	
	// copy the result array back from the GPU to the CPU
	// hipMemcpy( <dest_var> , <source_var> , <size> * sizeof( <type> ) , <direction> );
	

	
	
	// free the memory allocated on the CPU
	// free(<var>);

	
	// free the memory allocated on the GPU
	// hipFree(<var>);

	
	return 0; 
}
