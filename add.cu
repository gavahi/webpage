
#include <hip/hip_runtime.h>
#include "iostream"
#include "stdio.h"
#include <time.h>
#include <sys/time.h>

#define N 10000000

__global__ void add( int *a, int *b, int *c ) {
	int tid = blockIdx.x;  // handle the data at this index
	if (tid < N)
		c[tid] = a[tid] + b[tid]; 
}

int main( void ) {
	int *a, *b, *c;
	int *dev_a, *dev_b, *dev_c;
	//struct timeval t1,t2;
	
	hipEvent_t start, stop;
	
	  hipEventCreate( &start ) ;
	  hipEventCreate( &stop ) ;
	  
	
	
/* 	malloc( (void**)&a, N * sizeof(int) ) ;
	malloc( (void**)&b, N * sizeof(int) ) ;
	malloc( (void**)&c, N * sizeof(int) ) ; */
	
	a = (int*) malloc (N * sizeof(int));
	b = (int*) malloc (N * sizeof(int));
	c = (int*) malloc (N * sizeof(int));
	
	// allocate the memory on the GPU
	hipMalloc( (void**)&dev_a, N * sizeof(int) ) ;
	hipMalloc( (void**)&dev_b, N * sizeof(int) );
	hipMalloc( (void**)&dev_c, N * sizeof(int) );
	
	// fill the arrays 'a' and 'b' on the CPU
	for (int i=0; i<N; i++) {
		a[i] = i;
		b[i] = i * i;
	}
	
	// copy the arrays 'a' and 'b' to the GPU
	hipMemcpy( dev_a, a, N * sizeof(int),hipMemcpyHostToDevice );
	hipMemcpy( dev_b, b, N * sizeof(int),hipMemcpyHostToDevice );
	
	//gettimeofday(&t1,NULL);
	
	hipEventRecord( start, 0 ) ;
	
	add<<<N/128,128>>>( dev_a, dev_b, dev_c );
	hipMemcpy( c, dev_c, N * sizeof(int),hipMemcpyDeviceToHost );
	
	hipEventRecord( stop, 0 ) ;
	hipEventSynchronize( stop ) ;
	float  elapsedTime;
	hipEventElapsedTime( &elapsedTime, start, stop ) ;
	printf( "\n\n\tTime to generate: %3.1f ms\n\n", elapsedTime );
	
	
	//gettimeofday(&t2,NULL);	
	//double elapsed_time = ((t2.tv_sec - t1.tv_sec)*1000.0) + ((t2.tv_usec - t1.tv_usec)/1000.0);
	//printf("\n\n\tElapsed Time  = %.5f ms\n\n",elapsed_time);
	
	// copy the array 'c' back from the GPU to the CPU
	
	
	free( a );
	free( b );
	free( c );
	
	// free the memory allocated on the GPU
	hipFree( dev_a );
	hipFree( dev_b );
	hipFree( dev_c );
	
	return 0; 
}
